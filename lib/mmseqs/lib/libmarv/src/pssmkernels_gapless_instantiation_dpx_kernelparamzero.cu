#include "hip/hip_runtime.h"
#include "pssmkernels_gapless.cuh"

namespace cudasw4{

namespace kernelparamzero{

    #define ScoreOutputIterator TopNMaximaArray
    #define PositionsIterator decltype(thrust::make_counting_iterator<ReferenceIdT>(0))
    #define subjectIsCaseSensitive true
    #define X(g,r) \
        template void call_GaplessFilter_strided_PSSM_singletile_kernel<short2, 512, g, r, subjectIsCaseSensitive, ScoreOutputIterator, PositionsIterator>( \
            const char * const, \
            ScoreOutputIterator const, \
            const size_t* const, \
            const SequenceLengthT* const, \
            PositionsIterator const, \
            const int, \
            const SequenceLengthT, \
            const PSSM_2D_View<short2>&, \
            hipStream_t \
        );

        PSSM_GAPLESS_SINGLETILE_FOR_EACH_VALID_CONFIG_DO_X

    #undef X
    #undef subjectIsCaseSensitive
    #undef PositionsIterator
    #undef ScoreOutputIterator


    #define ScoreOutputIterator TopNMaximaArray
    #define PositionsIterator ReferenceIdT*
    #define subjectIsCaseSensitive true
    #define X(g,r) \
        template void call_GaplessFilter_strided_PSSM_singletile_kernel<short2, 512, g, r, subjectIsCaseSensitive, ScoreOutputIterator, PositionsIterator>( \
            const char * const, \
            ScoreOutputIterator const, \
            const size_t* const, \
            const SequenceLengthT* const, \
            PositionsIterator const, \
            const int, \
            const SequenceLengthT, \
            const PSSM_2D_View<short2>&, \
            hipStream_t \
        );

        PSSM_GAPLESS_SINGLETILE_FOR_EACH_VALID_CONFIG_DO_X

    #undef X
    #undef subjectIsCaseSensitive
    #undef PositionsIterator
    #undef ScoreOutputIterator


    #define ScoreOutputIterator TopNMaximaArray
    #define PositionsIterator decltype(thrust::make_counting_iterator<ReferenceIdT>(0))
    #define subjectIsCaseSensitive true
    #define X(g,r) \
        template void call_GaplessFilter_strided_PSSM_multitile_kernel<short2, 512, g, r, subjectIsCaseSensitive, ScoreOutputIterator, PositionsIterator>( \
            int, \
            const char * const, \
            ScoreOutputIterator const, \
            const size_t* const, \
            const SequenceLengthT* const, \
            PositionsIterator const, \
            const int, \
            const SequenceLengthT, \
            const PSSM_2D_View<short2>&, \
            float2*, \
            size_t, \
            hipStream_t \
        );

        PSSM_GAPLESS_MULTITILE_FOR_EACH_VALID_CONFIG_DO_X

    #undef X
    #undef subjectIsCaseSensitive
    #undef PositionsIterator
    #undef ScoreOutputIterator


    #define ScoreOutputIterator TopNMaximaArray
    #define PositionsIterator ReferenceIdT*
    #define subjectIsCaseSensitive true
    #define X(g,r) \
        template void call_GaplessFilter_strided_PSSM_multitile_kernel<short2, 512, g, r, subjectIsCaseSensitive, ScoreOutputIterator, PositionsIterator>( \
            int, \
            const char * const, \
            ScoreOutputIterator const, \
            const size_t* const, \
            const SequenceLengthT* const, \
            PositionsIterator const, \
            const int, \
            const SequenceLengthT, \
            const PSSM_2D_View<short2>&, \
            float2*, \
            size_t, \
            hipStream_t \
        );

        PSSM_GAPLESS_MULTITILE_FOR_EACH_VALID_CONFIG_DO_X

    #undef X
    #undef subjectIsCaseSensitive
    #undef PositionsIterator
    #undef ScoreOutputIterator

} //namespace hardcodedzero


} //namespace cudasw4